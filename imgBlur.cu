#include "hip/hip_runtime.h"
#include "libwb/wb.h"
#include "my_timer.h"

#define wbCheck(stmt)							\
  do {									\
    hipError_t err = stmt;						\
    if (err != hipSuccess) {						\
      wbLog(ERROR, "Failed to run stmt ", #stmt);			\
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));	\
      return -1;							\
    }									\
  } while (0)

#define BLUR_SIZE 21

///////////////////////////////////////////////////////
//@@ INSERT YOUR CODE HERE
__global__ void blurKernel(float *out, float *in, int width, int height) 
{
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int BLUR_SIZE = 2;
  
  if (Col < w && Row < h) 
  {
    int pixVal = 0; int pixels = 0;
    // Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
    for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) 
    {
      for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) 
      {
        int curRow = Row + blurRow;
        int curCol = Col + blurCol;
        // Verify we have a valid image pixel
        if(curRow > -1 && curRow < h && curCol > -1 && curCol < w) 
        {
          pixVal += in[curRow * w + curCol];
          // Keep track of number of pixels in the accumulated total
          pixels++;
        }
      }
    }
    // Write our new average pixel value out
    out[Row * w + Col] = (unsigned char)(pixVal / pixels);
  }
}
///////////////////////////////////////////////////////

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  wbImage_t goldImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *goldOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);
  inputImage = wbImport(inputImageFile);

  char *goldImageFile = argv[2];
  goldImage = wbImport(goldImageFile);

  // The input image is in grayscale, so the number of channels is 1
  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  // Get host input and output image data
  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);
  goldOutputImageData = wbImage_getData(goldImage);

  // Start timer
  timespec timer = tic();
  
  ////////////////////////////////////////////////
  //@@ INSERT AND UPDATE YOUR CODE HERE

  // Allocate cuda memory for device input and ouput image data
  hipMalloc((void **)&deviceInputImageData, imageWidth * imageHeight * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData, imageWidth * imageHeight * sizeof(float));

  // Transfer data from CPU to GPU
  hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * sizeof(float), hipMemcpyHostToDevice);
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid(ceil(imgWidth/16.0), ceil(imgHeight/16.0), 1);
  
  // Call your GPU kernel 10 times
  for(int i = 0; i < 10; i++)
  imgBlurGPU<<<dimGrid, dimBlock>>>(deviceOutputImageData, deviceInputImageData, imageWidth, imageHeight);

  // Transfer data from GPU to CPU
  hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
  ///////////////////////////////////////////////////////
  
  // Stop and print timer
  toc(&timer, "GPU execution time (including data transfer) in seconds");

  // Check the correctness of your solution
  //wbSolution(args, outputImage);

   for(int i=0; i<imageHeight; i++){
     for(int j=0; j<imageWidth; j++){
       if(abs(hostOutputImageData[i*imageWidth+j]-goldOutputImageData[i*imageWidth+j])/goldOutputImageData[i*imageWidth+j]>0.01){
          printf("Incorrect output image at pixel (%d, %d): goldOutputImage = %f, hostOutputImage = %f\n", i, j, goldOutputImageData[i*imageWidth+j],hostOutputImageData[i*imageWidth+j]);
	        return -1;
       }
     }
   }
   printf("Correct output image!\n");

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
